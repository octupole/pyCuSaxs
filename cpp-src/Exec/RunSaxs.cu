#include "hip/hip_runtime.h"
#include "RunSaxs.h"
#include <fstream>
#include "fmt/core.h"
#include "fmt/format.h"
#include "Options.h"
#include "saxsKernel.h"
#include "Cell.h"

/// Creates a vector of integers with a specified start, end, and step.
///
/// This function calculates the size of the vector based on the given start, end, and step values.
/// It then creates a vector and fills it with sequential integers starting from 0, and transforms
/// the values to match the desired sequence.
///
/// @param start The starting value for the sequence.
/// @param end The ending value for the sequence.
/// @param step The step size between values.
/// @return A vector of integers representing the desired sequence.
std::vector<int> RunSaxs::createVector(int start, int end, int step)
{
    // Calculate the size of the vector
    int size = (end - start) / step + 1;

    // Create a vector to hold the values
    std::vector<int> result(size);

    // Fill the vector with sequential integers starting from 0
    std::iota(result.begin(), result.end(), 0);

    // Transform the values to match the desired sequence
    std::transform(result.begin(), result.end(), result.begin(),
                   [start, step](int x)
                   { return start + x * step; });

    return result;
}
/// Runs the SAXS (Small-Angle X-ray Scattering) analysis on a range of frames.
///
/// This function creates a vector of frame indices to process, and then iterates over each frame.
/// For each frame, it retrieves the centered coordinates and box dimensions, calculates the
/// transformation matrices, and runs the SAXS kernel on the coordinates. The elapsed time for
/// the entire process is measured and printed.
///
/// @param beg The starting frame index.
/// @param end The ending frame index.
/// @param dt The step size between frames.

bool RunSaxs::loadFrameData(py::handle frame_handle, FrameData &data)
{
    try
    {
        py::dict frame_data = py::cast<py::dict>(frame_handle);

        data.frame_num = frame_data["frame"].cast<int>();
        data.time = frame_data["time"].cast<float>();

        // Extract positions array
        py::array_t<float> positions = frame_data["positions"].cast<py::array_t<float>>();
        auto pos = positions.unchecked<2>();

        const size_t n_atoms = pos.shape(0);

        // Resize and fill coords in std::vector<std::vector<float>> format
        data.coords.resize(n_atoms, std::vector<float>(3));
        for (size_t i = 0; i < n_atoms; ++i)
        {
            data.coords[i][0] = pos(i, 0);
            data.coords[i][1] = pos(i, 1);
            data.coords[i][2] = pos(i, 2);
        }

        // Extract box dimensions
        py::array_t<float> box_array = frame_data["box"].cast<py::array_t<float>>();
        auto box_data = box_array.unchecked<2>();

        data.box.resize(3, std::vector<float>(3));
        for (size_t i = 0; i < 3; ++i)
        {
            for (size_t j = 0; j < 3; ++j)
            {
                data.box[i][j] = box_data(i, j);
            }
        }

        return true;
    }
    catch (const std::exception &e)
    {
        std::cerr << "Error loading frame data: " << e.what() << std::endl;
        return false;
    }
}

void RunSaxs::Run(py::object Topol, int beg, int end, int dt)
{
    const int start_frame = beg;
    const int stop_frame = end;
    const int stride = std::max(dt, 1);

    py::object analyzer = std::move(Topol);

    try
    {
        // ===== Setup phase (with GIL) =====
        py::gil_scoped_acquire gil;

        // Get atom index map from Python
        std::map<std::string, std::vector<int>> index_map;
        py::dict gather_dict = analyzer.attr("get_atom_index")();
        for (auto item : gather_dict)
        {
            std::string key = py::str(item.first);
            std::vector<int> value = item.second.cast<std::vector<int>>();
            index_map[key] = value;
        }

        // Initialize SAXS kernel
        auto start = std::chrono::high_resolution_clock::now();
        saxsKernel myKernel(Options::nx, Options::ny, Options::nz, Options::order);
        myKernel.setnpx(8);
        myKernel.scaledCell();

        // Read first frame to get box dimensions for initialization
        analyzer.attr("read_frame")(0);
        auto box_dimensions = analyzer.attr("get_box")().cast<std::vector<std::vector<float>>>();
        Cell::calculateMatrices(box_dimensions);
        auto oc = Cell::getOC();

        // Setup padding if needed
        if (Options::myPadding == padding::given)
        {
            if (index_map.find("Na") != index_map.end() && Options::Sodium == 0)
                Options::Sodium = index_map["Na"].size();
            if (index_map.find("Cl") != index_map.end() && Options::Chlorine == 0)
                Options::Chlorine = index_map["Cl"].size();

            AtomCounter Density(box_dimensions[0][0], box_dimensions[1][1],
                                box_dimensions[2][2], Options::Sodium, Options::Chlorine,
                                Options::Wmodel, Options::nx, Options::ny, Options::nz);
            Options::myWmodel = Density.calculateAtomCounts();
            for (auto &pair : Options::myWmodel)
            {
                auto type = pair.first;
                if (index_map.find(type) == index_map.end())
                    pair.second = 0.0f;
            }
        }

        // Finalize kernel setup
        myKernel.resetHistogramParameters(oc);
        myKernel.createMemory();
        myKernel.writeBanner();
        myKernel.setcufftPlan(Options::nnx, Options::nny, Options::nnz);

        // ===== Streaming phase =====
        // Create iterator with GIL held
        auto frames_iter = analyzer.attr("iter_frames_stream")(
            start_frame, stop_frame + 1, stride);

        // Double buffering for pipeline optimization
        FrameData current_frame;
        FrameData next_frame;
        bool has_next = false;

        // Prime the pipeline - load first frame
        auto iter = frames_iter.begin();
        if (iter != frames_iter.end())
        {
            has_next = loadFrameData(*iter, next_frame);
            ++iter;
        }

        while (has_next)
        {
            // Swap buffers (cheap pointer swap)
            std::swap(current_frame, next_frame);

            // Start loading next frame while processing current
            bool has_more = (iter != frames_iter.end());
            if (has_more)
            {
                has_next = loadFrameData(*iter, next_frame);
                ++iter;
            }
            else
            {
                has_next = false;
            }

            // Release GIL for GPU processing
            {
                py::gil_scoped_release release;

                // ===== Process current frame (GIL released) =====
                try
                {
                    // Calculate transformation matrices from box dimensions
                    Cell::calculateMatrices(current_frame.box);
                    auto co = Cell::getCO();
                    auto oc = Cell::getOC();

                    // Run SAXS kernel computation
                    myKernel.runPKernel(current_frame.frame_num, current_frame.time,
                                        current_frame.coords, index_map, oc);
                }
                catch (const std::exception &e)
                {
                    std::cerr << "Error processing frame " << current_frame.frame_num
                              << ": " << e.what() << std::endl;
                }
                // ================================================
            }
            // GIL automatically re-acquired here when 'release' goes out of scope
        }

        // ===== Finalization phase (with GIL) =====
        std::vector<std::vector<double>> myhisto;

        if (Options::Simulation == "nvt")
        {
            myKernel.getHistogram(oc);
        }
        myhisto = myKernel.getSaxs();

        // Write results
        std::ofstream myfile;
        myfile.open(Options::outFile);
        for (auto data : myhisto)
        {
            myfile << std::fixed << std::setw(10) << std::setprecision(5) << data[0];
            myfile << std::scientific << std::setprecision(5) << std::setw(12) << data[1] << std::endl;
        }
        myfile.close();

        // Print timing information
        auto frames_to_process = createVector(start_frame, stop_frame, stride);
        std::cout << "Done " << frames_to_process.size() << " Steps" << std::endl;
        std::cout << "Results written to " << Options::outFile << std::endl;

        auto end0 = std::chrono::high_resolution_clock::now();
        auto duration_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end0 - start);
        auto cudaTime = myKernel.getCudaTime();
        auto totalTime = duration_ms.count() / (float)frames_to_process.size();
        auto readTime = totalTime - cudaTime;

        std::string banner = fmt::format(
            "\n=========================================================\n"
            "=                                                       =\n"
            "=                    CuSAXS Timing                     =\n"
            "=                                                       =\n"
            "=           CUDA Time:     {:<10.2f} ms/per step       =\n"
            "=           Read Time:     {:<10.2f} ms/per step       =\n"
            "=           Total Time:    {:<10.2f} ms/per step       =\n"
            "=                                                       =\n"
            "=========================================================\n\n",
            cudaTime, readTime, totalTime);

        fmt::print("{}", banner);
    }
    catch (const py::error_already_set &err)
    {
        std::cerr << "Python error while iterating frames: " << err.what() << std::endl;
    }
};

RunSaxs::~RunSaxs() {};
